#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "CompareImageCUDA.h"

__global__ void CompareCUDA(unsigned char* img, int channels);

void ImageCompareCUDA(unsigned char* inputImage, int height, int width, int channels) {
	unsigned char* devInputImage = NULL;
	hipMalloc((void**)&devInputImage, height * width * channels);

	hipMemcpy(devInputImage, inputImage, height * width * channels, hipMemcpyHostToDevice);
	dim3 gridImage(width , height);

	CompareCUDA << <gridImage,1  >> > (devInputImage, channels);


	hipMemcpy(inputImage, devInputImage, height * width * channels, hipMemcpyDeviceToHost);

	hipFree(devInputImage);
}

__global__ void CompareCUDA(unsigned char* img, int channels) {
	// int id = (threadIdx.x + blockIdx.x * blockDim.x) * channels;
	int x = blockIdx.x;
	int y = blockIdx.y;
	int id = (x + y * gridDim.x) * channels;


	for (int i = 0; i < channels; i++) {
		img[id + i] = 255 - img[id + i];
	}

}