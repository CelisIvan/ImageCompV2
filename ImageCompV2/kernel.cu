#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include "CompareImageCUDA.h"

#include <cstdlib>

__global__ void compare_CUDA(unsigned char* image, unsigned char* image2, unsigned char* image3, int channels, int option);

void Image_Comparisson_CUDA(unsigned char* inputImage, unsigned char* inputImage2, unsigned char* diffImage, int height, int width, int channels, int option) {
	unsigned char* Dev_Input_Image = NULL;
	unsigned char* Dev_Input_Image2 = NULL;
	unsigned char* Dev_outputImage = NULL;
	//allocate
	hipMalloc((void**)&Dev_Input_Image, height * width * channels);
	hipMalloc((void**)&Dev_Input_Image2, height * width * channels);
	hipMalloc((void**)&Dev_outputImage, height * width * channels);

	//copy data from cpu to gpu
	hipMemcpy(Dev_Input_Image, inputImage, height * width * channels, hipMemcpyHostToDevice);
	hipMemcpy(Dev_Input_Image2, inputImage2, height * width * channels, hipMemcpyHostToDevice);
	hipMemcpy(Dev_outputImage, diffImage, height * width * channels, hipMemcpyHostToDevice);


	dim3 gridImage(width * height / 512);

	// compare_CUDA << <gridImage, 3 >> > (Dev_Input_Image, Dev_Input_Image2, Dev_outputImage, channels);
	compare_CUDA << <gridImage, 512 >> > (Dev_Input_Image, Dev_Input_Image2, Dev_outputImage, channels, option);


	hipMemcpy(inputImage, Dev_Input_Image, height * width * channels, hipMemcpyDeviceToHost);
	hipMemcpy(inputImage2, Dev_Input_Image2, height * width * channels, hipMemcpyDeviceToHost);
	hipMemcpy(diffImage, Dev_outputImage, height * width * channels, hipMemcpyDeviceToHost);

	hipFree(Dev_Input_Image);
	hipFree(Dev_Input_Image2);
	hipFree(Dev_outputImage);
}

__global__ void compare_CUDA(unsigned char* image, unsigned char* image2, unsigned char* image3, int channels, int option) {
	//int y = blockIdx.y;
	int id = (threadIdx.x + blockIdx.x * blockDim.x) * channels;

	//col = ThreadIdx.x + BlockIdx.x * BlockDim.x;
	//row = BlockIdx.y;

	if (option == 1) {
		for (int i = 0; i < channels; i++) {
			if (image[id + i] != image2[id + i]) {
				image3[id + i] = 255 - image[id + i];
			}
			else {
				image3[id + i] = 0;
			}

		}
	}
	else {
		for (int i = 0; i < channels; i++) {
			int dif = abs(image[id + i] - image2[id + i]);
			if (dif > 90) {
				image3[id + i] = 255 - image[id + i];
			}
			else {
				image3[id + i] = 0;
			}
		}
	}
}
